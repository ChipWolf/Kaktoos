
#include <hip/hip_runtime.h>
#include <cstdint>
#include <memory.h>
#include <cstdio>
#include <ctime>
#include <thread>
#include <vector>
#include <mutex>
#include <chrono>

#define RANDOM_MULTIPLIER 0x5DEECE66DULL
#define RANDOM_ADDEND 0xBULL
#define RANDOM_MASK ((1ULL << 48ULL) - 1ULL)

#ifndef FLOOR_LEVEL
#define FLOOR_LEVEL 63LL
#endif

#ifndef WANTED_CACTUS_HEIGHT
#define WANTED_CACTUS_HEIGHT 8ULL
#endif

#ifndef WORK_UNIT_SIZE
#define WORK_UNIT_SIZE (1ULL << 23ULL)
#endif

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 1024ULL
#endif

#ifndef GPU_COUNT
#define GPU_COUNT 1ULL
#endif

#ifndef OFFSET
#define OFFSET 0
#endif

#ifndef END
#define END (1ULL << 48ULL)
#endif

__device__ inline int8_t extract(const int8_t heightMap[], uint32_t i) {
    return (int8_t) (heightMap[i >> 1ULL] >> ((i & 1ULL) << 2ULL)) & 0xF;
}

__device__ inline void increase(int8_t heightMap[], uint32_t i) {
    heightMap[i >> 1ULL] += 1ULL << ((i & 1ULL) << 2ULL);
}

namespace java_random {

    // Random::next(bits)
    __device__ inline uint32_t next(uint64_t *random, int32_t bits) {
        *random = (*random * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
        return (uint32_t) (*random >> (48ULL - bits));
    }

    __device__ inline int32_t next_int_unknown(uint64_t *seed, int16_t bound) {
        if ((bound & -bound) == bound) {
            *seed = (*seed * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
            return (int32_t) ((bound * (*seed >> 17ULL)) >> 31ULL);
        }

        int32_t bits, value;
        do {
            *seed = (*seed * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
            bits = *seed >> 17ULL;
            value = bits % bound;
        } while (bits - value + (bound - 1) < 0);
        return value;
    }

    // Random::nextInt(bound)
    __device__ inline uint32_t next_int(uint64_t *random) {
        return java_random::next(random, 31) % 3;
    }

}

__global__ __launch_bounds__(BLOCK_SIZE, 2) void crack(uint64_t seed_offset, int32_t *num_seeds, uint64_t *seeds) {
    uint64_t originalSeed = blockIdx.x * blockDim.x + threadIdx.x + seed_offset;
    uint64_t seed = originalSeed;

    int8_t heightMap[512];

#pragma unroll
    for (int i = 0; i < 512; i++) {
        heightMap[i] = 0;
    }

    int16_t currentHighestPos = 0;
    int16_t terrainHeight;
    int16_t initialPosX, initialPosY, initialPosZ;
    int16_t posX, posY, posZ;
    int16_t offset, posMap;

    int16_t i, a, j;

    for (i = 0; i < 10; i++) {
        // Keep, most threads finish early this way
        if (WANTED_CACTUS_HEIGHT - extract(heightMap, currentHighestPos) > 9 * (10 - i))
            return;

        initialPosX = java_random::next(&seed, 4) + 8;
        initialPosZ = java_random::next(&seed, 4) + 8;
        terrainHeight = (extract(heightMap, initialPosX + initialPosZ * 32) + FLOOR_LEVEL + 1) * 2;

        initialPosY = java_random::next_int_unknown(&seed, terrainHeight);

        for (a = 0; a < 10; a++) {
            posX = initialPosX + java_random::next(&seed, 3) - java_random::next(&seed, 3);
            posY = initialPosY + java_random::next(&seed, 2) - java_random::next(&seed, 2);
            posZ = initialPosZ + java_random::next(&seed, 3) - java_random::next(&seed, 3);

            posMap = posX + posZ * 32;
            // Keep
            if (posY <= extract(heightMap, posMap) + FLOOR_LEVEL && posY >= 0)
                continue;

            offset = 1 + java_random::next_int_unknown(&seed, java_random::next_int(&seed) + 1);

            for (j = 0; j < offset; j++) {
                if ((posY + j - 1) > extract(heightMap, posMap) + FLOOR_LEVEL || posY < 0) continue;
                if ((posY + j) <= extract(heightMap, (posX + 1) + posZ * 32) + FLOOR_LEVEL && posY >= 0) continue;
                if ((posY + j) <= extract(heightMap, posX + (posZ - 1) * 32) + FLOOR_LEVEL && posY >= 0) continue;
                if ((posY + j) <= extract(heightMap, (posX - 1) + posZ * 32) + FLOOR_LEVEL && posY >= 0) continue;
                if ((posY + j) <= extract(heightMap, posX + (posZ + 1) * 32) + FLOOR_LEVEL && posY >= 0) continue;

                increase(heightMap, posMap);

                if (extract(heightMap, currentHighestPos) < extract(heightMap, posMap)) {
                    currentHighestPos = posMap;
                }
            }
        }

        if (extract(heightMap, currentHighestPos) >= WANTED_CACTUS_HEIGHT) {
            seeds[atomicAdd(num_seeds, 1)] = originalSeed;
            return;
        }
    }
}


struct GPU_Node {
    int *num_seeds;
    uint64_t *seeds;
};

void setup_gpu_node(GPU_Node *node, int32_t gpu) {
    hipSetDevice(gpu);
    hipMallocManaged(&node->num_seeds, sizeof(*node->num_seeds));
    hipMallocManaged(&node->seeds, 1ULL << 10ULL);
}

GPU_Node nodes[GPU_COUNT];
uint64_t offset = OFFSET;
uint64_t count = 0;
std::mutex info_lock;
std::vector<uint64_t> seeds;

void gpu_manager(int32_t gpu_index) {
    std::string fileName = "kaktoos_seeds" + std::to_string(gpu_index) + ".txt";
    FILE *out_file = fopen(fileName.c_str(), "w");
    hipSetDevice(gpu_index);
    while (offset < END) {
        *nodes[gpu_index].num_seeds = 0;
        crack<<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE, 0>>>(offset, nodes[gpu_index].num_seeds,
                                                              nodes[gpu_index].seeds);
        info_lock.lock();
        offset += WORK_UNIT_SIZE;
        info_lock.unlock();
        hipDeviceSynchronize();
        for (int32_t i = 0, e = *nodes[gpu_index].num_seeds; i < e; i++) {
            fprintf(out_file, "%lld\n", (long long int) nodes[gpu_index].seeds[i]);
            seeds.push_back(nodes[gpu_index].seeds[i]);
        }
        fflush(out_file);
        info_lock.lock();
        count += *nodes[gpu_index].num_seeds;
        info_lock.unlock();
    }
    fclose(out_file);
}

int main() {
    printf("Searching %ld total seeds...\n", (long int) (END - OFFSET));

    std::thread threads[GPU_COUNT];

    time_t startTime = time(nullptr), currentTime;
    for (int32_t i = 0; i < GPU_COUNT; i++) {
        setup_gpu_node(&nodes[i], i);
        threads[i] = std::thread(gpu_manager, i);
    }

    using namespace std::chrono_literals;

    while (offset < END) {
        time(&currentTime);
        int timeElapsed = (int) (currentTime - startTime);
        double speed = (double) (offset - OFFSET) / (double) timeElapsed / 1000000.0;
        printf("Searched %lld seeds, offset: %lld found %lld matches. Time elapsed: %ds. Speed: %.2fm seeds/s. %f%%\n",
               (long long int) (offset - OFFSET),
               (long long int) offset,
               (long long int) count,
               timeElapsed,
               speed,
               (double) (offset - OFFSET) / (END - OFFSET) * 100);

        if (timeElapsed % 2000 == 0) {
            printf("Backup seed list:\n");
            for (auto &seed : seeds) {
                printf("%llu\n", (unsigned long long) seed);
            }
        }

        std::this_thread::sleep_for(1s);
    }

    for (auto &thread : threads) {
        thread.join();
    }

    printf("Done!\n");
    printf("But, verily, it be the nature of dreams to end.\n");

}